
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void kernel(short *a, int offset)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  float x = (float)i;
  float s = sinf(x);
  float c = cosf(x);
  a[i] = a[i] + sqrtf(s*s+c*c);
}

float maxError(short *a, int n)
{
  float maxE = 0;
  for (int i = 0; i < n; i++) {
    float error = fabs(a[i]-1.0f);
    if (error > maxE) maxE = error;
  }
  return maxE;
}

int main(int argc, char **argv)
{
  const int n = 2296960*100;
  const int blockSize = 1024, nStreams = atoi(argv[1]);
  const int streamSize = n / nStreams;
  const int streamBytes = streamSize * sizeof(short);
  const int bytes = n * sizeof(short);

  printf("Running with nStreams: %d\n", nStreams);

  int devId = 0;
  if (argc > 2) devId = atoi(argv[2]);
  
  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );

  // allocate pinned host memory and device memory
  short *a, *d_a;
  checkCuda( hipHostMalloc((void**)&a, bytes, hipHostMallocDefault) ); // host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); // device  
  
  float ms; // elapsed time in milliseconds

  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );

  // baseline case - sequential transfer and execute
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent,0) );
  checkCuda( hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice) );
  kernel<<<n/blockSize, blockSize>>>(d_a, 0);
  checkCuda( hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for sequential trasnfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));
  
  // asynchronous version 1: loop over {copy, kernel, copy}
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent, 0) );
  hipProfilerStart();
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
    kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }
  hipProfilerStop();
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));
  
  // asynchronous version 2:
  // loop over copy, loop over kernel, loop over copy
  memset(a, 0, bytes);
  checkCuda( hipEventRecord(startEvent, 0) );
  hipProfilerStart();
  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
  }
  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
  }
  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }
  hipProfilerStop();
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V2 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamDestroy(stream[i]) );
  hipFree(d_a);
  hipHostFree(a);

  return 0;
}
