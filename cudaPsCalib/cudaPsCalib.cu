
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

// TITAN X Spec
#define BLOCK_D1 1024
#define BLOCK_D2 1
#define BLOCK_D3 1

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void kernel(short *a, int offset, short *dark, int offsetDark, int *sectorSum)
{
  // note that this assumes no third dimension to the grid
  // id of the block
  int myblock = blockIdx.x + blockIdx.y * gridDim.x;
  // size of each block (within grid of blocks)
  int blocksize = blockDim.x * blockDim.y * blockDim.z;
  // id of thread in a given block
  int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
  // assign overall id/index of the thread
  int idx = myblock * blocksize + subthread;

  a[idx + offset] -= 1;  
  // calculate sum per sector
  //int mySector = threadIdx.z + (offset / (blockDim.x * blockDim.y)); 
  //atomicAdd(&sectorSum[mySector], mySector);
  //sectorSum[mySector] = mySector;
}

__global__ void common_mode(int *blockSum, int *sectorMean, int offsetSector)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  // calculate sector mean
  atomicAdd(&sectorMean[offsetSector], blockSum[i]);
}

__global__ void common_mode_apply(short *a, int offset, int *sectorMean, int offsetSector, int sectorSize)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  a[i] = a[i] - (sectorMean[offsetSector]/sectorSize);
}

/* ---------------------- host code -----------------------------*/
void fill( short *p, int n, int val ) {
  for(int i = 0; i < n; i++){
    p[i] = val;
  }
}

float maxError(short *a, int n)
{
  float maxE = 0;
  for (int i = 0; i < n; i++) {
    float error = fabs(a[i]-1.0f);
    if (error > maxE) maxE = error;
  }
  return maxE;
}

int main(int argc, char **argv)
{
  const int maxQuads = 4, maxSectors = 8;
  const int nColumns = 185, nRows = 388;
  const int nPixels = nColumns * nRows * maxSectors * maxQuads;
  const int nEvents = atoi(argv[1]);
  const int n = nPixels * nEvents;

  const int nStreams = atoi(argv[2]);
  const int streamSize = ceil(n / nStreams);
  const int nSectors = maxQuads * maxSectors * nEvents;

  const int streamBytes = streamSize * sizeof(short);
  const int bytes = n * sizeof(short);
  const int darkBytes = nPixels * sizeof(short);
  const int sumSectorBytes = nSectors * sizeof(int);

  // a block has 1024 threads
  const dim3 blockSize(BLOCK_D1, BLOCK_D2, BLOCK_D3);
  printf("Block dimension is %i x %i x %i\n", BLOCK_D1, BLOCK_D2, BLOCK_D3);  

  // determine number of blocks we need for a given problem size
  int tmp = ceil(pow(n/(BLOCK_D1 * BLOCK_D2 * BLOCK_D3 * nStreams), 0.5));
  printf("Grid dimension is %i x %i\n", tmp, tmp);
  dim3 gridSize(tmp, tmp, 1);

  int nthreads = BLOCK_D1*BLOCK_D2*BLOCK_D3*tmp*tmp;
  if (nthreads < n){
    printf("\n================ NOT ENOUGH THREADS TO COVER N=%d =======================\n\n", n);
  } else {
    printf("Launching %d threads (N=%d)\n", nthreads, n);

  }
 
  printf("Running with nStreams: %d streamSize: %d\n", nStreams, streamSize);

  int devId = 0;
  if (argc > 3) devId = atoi(argv[3]);
  
  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );

  // allocate pinned host memory and device memory
  short *a, *d_a; // data
  checkCuda( hipHostMalloc((void**)&a, bytes, hipHostMallocDefault) ); // host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); // device  
  short *dark, *d_dark; // dark
  checkCuda( hipHostMalloc((void**)&dark, darkBytes, hipHostMallocDefault) ); 
  checkCuda( hipMalloc((void**)&d_dark, darkBytes) ); 
  int *d_sectorSum, *sectorSum; // sum of each sector
  checkCuda( hipMalloc((void**)&d_sectorSum, sumSectorBytes) ); 
  hipMemset(d_sectorSum, 0, sumSectorBytes);
  sectorSum = (int *) malloc(sumSectorBytes);
  
  // prepare raw and dark data
  fill(a, n, 3);
  fill(dark, nPixels, 1);
  printf("Input values (Data): %d %d %d...%d %d %d\n", a[0], a[1], a[2], a[n-3], a[n-2], a[n-1]);
  printf("Input values (Dark): %d %d %d...%d %d %d\n", dark[0], dark[1], dark[2], dark[nPixels-3], dark[nPixels-2], dark[nPixels-1]);

  // serial copy for one dark 
  checkCuda( hipMemcpy(d_dark, dark, darkBytes, hipMemcpyHostToDevice) );

  float ms; // elapsed time in milliseconds

  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );

  // asynchronous version 1: loop over {copy, kernel, copy}
  checkCuda( hipEventRecord(startEvent, 0) );
  hipProfilerStart();
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    int offsetDark = offset % nPixels;
    printf("Stream :%d offset:%d offsetDark:%d\n", i, offset, offsetDark);
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
    kernel<<<gridSize, blockSize, 0, stream[i]>>>(d_a, offset, d_dark, offsetDark, d_sectorSum);
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }
  
  //cudaMemcpy(sectorSum, d_sectorSum, sumSectorBytes, cudaMemcpyDeviceToHost);
  //for (int i =0; i< 10; i++){
  //  printf("i: %d, sectorSum[i]: %d \n", i, sectorSum[i]);
  //}
  printf("Output values: %d %d %d...%d %d %d\n", a[0], a[1], a[2], a[n-3], a[n-2], a[n-1]);
  hipProfilerStop();
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));
  
  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamDestroy(stream[i]) );
  hipFree(d_a);
  hipHostFree(a);
  //cudaFree(d_dark);
  //cudaFreeHost(dark);

  return 0;
}