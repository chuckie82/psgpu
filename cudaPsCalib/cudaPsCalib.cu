
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define N_PIXELS 2296960
#define SECTOR_SIZE 71780

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void kernel(short *a, int offset, short *dark, int *blockSum)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  int iDark = i % N_PIXELS;
  a[i] -= dark[iDark];
  
  // calculate sum per block
  int iBlock = floor( (double) i / blockDim.x );
  atomicAdd(&blockSum[iBlock], a[i]);
}

__global__ void common_mode(int *blockSum, int offset, int *sectorSum)
{
  int i = offset + threadIdx.x + blockIdx.x * blockDim.x;

  // calculate sector sum
  int iSector = floor( (double) i / blockDim.x );
  atomicAdd(&sectorSum[iSector], blockSum[i]);
}

__global__ void common_mode_apply(short *a, int offset, int *sectorSum)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  int iSector = floor( (double) i / SECTOR_SIZE );
  a[i] = a[i] - (sectorSum[iSector] / SECTOR_SIZE);
}

/* ---------------------- host code -----------------------------*/
void fill( short *p, int n, int val ) {
  for(int i = 0; i < n; i++){
    p[i] = val;
  }
}

float maxError(short *a, int n)
{
  float maxE = 0;
  for (int i = 0; i < n; i++) {
    float error = fabs(a[i]-0.0f);
    if (error > maxE) maxE = error;
  }
  return maxE;
}

int main(int argc, char **argv)
{
  const int maxQuads = 4, maxSectors = 8;
  const int nColumns = 185, nRows = 388;
  const int nPixels = nColumns * nRows * maxSectors * maxQuads;
  const int nEvents = atoi(argv[1]);
  const int n = nPixels * nEvents;

  int nStreams = 32;
  if (argc > 2) nStreams = atoi(argv[2]);
  const int streamSize = n / nStreams;
  const int streamBytes = streamSize * sizeof(short);
  
  const int bytes = n * sizeof(short);
  
  const int darkBytes = nPixels * sizeof(short);
  
  const int blockSize = 185;
  const int nBlocks = n / blockSize;
  const int blockSumBytes = nBlocks * sizeof(int);
  
  const int nSectors = nBlocks / nRows;   
  const int sectorSumBytes = nSectors * sizeof(int);

  printf("Running with nStreams: %d streamSize: %d\n", nStreams, streamSize);
  int gridSize = streamSize / blockSize;
  printf("blockSize: %d gridSize: %d\n", blockSize, gridSize);
  
  int devId = 0;
  if (argc > 3) devId = atoi(argv[3]);
  
  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );

  // allocate pinned host memory and device memory
  short *a, *d_a; // data
  checkCuda( hipHostMalloc((void**)&a, bytes, hipHostMallocDefault) ); // host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); // device  
  
  short *dark, *d_dark; // dark
  checkCuda( hipHostMalloc((void**)&dark, darkBytes, hipHostMallocDefault) ); 
  checkCuda( hipMalloc((void**)&d_dark, darkBytes) ); 
  
  int *d_blockSum; // sum of each block
  checkCuda( hipMalloc((void**)&d_blockSum, blockSumBytes) ); 
  hipMemset(d_blockSum, 0, blockSumBytes);
  
  int *d_sectorSum; // sum of each sector
  checkCuda( hipMalloc((void**)&d_sectorSum, sectorSumBytes) );
  hipMemset(d_sectorSum, 0, sectorSumBytes);
  
  // prepare raw and dark data
  fill(a, n, 2);
  fill(dark, nPixels, 1);
  printf("Input values (Data): %d %d %d...%d %d %d\n", a[0], a[1], a[2], a[n-3], a[n-2], a[n-1]);
  printf("Input values (Dark): %d %d %d...%d %d %d\n", dark[0], dark[1], dark[2], dark[nPixels-3], dark[nPixels-2], dark[nPixels-1]);

  // serial copy for one dark 
  checkCuda( hipMemcpy(d_dark, dark, darkBytes, hipMemcpyHostToDevice) );

  float ms; // elapsed time in milliseconds

  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );

  // asynchronous version 1: loop over {copy, kernel, copy}
  checkCuda( hipEventRecord(startEvent, 0) );
  hipProfilerStart();
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    int offsetSector = i * (streamSize / blockSize);
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
    kernel<<<gridSize, blockSize, 0, stream[i]>>>(d_a, offset, d_dark, d_blockSum);
    common_mode<<<nBlocks/(nStreams * nRows), nRows, 0, stream[i]>>>(d_blockSum, offsetSector, d_sectorSum); 
    common_mode_apply<<<gridSize, blockSize, 0, stream[i]>>>(d_a, offset, d_sectorSum);
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }
  hipProfilerStop(); 
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));
  
  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamDestroy(stream[i]) );
  hipFree(d_a);
  hipHostFree(a);
  //cudaFree(d_dark);
  //cudaFreeHost(dark);

  return 0;
}
