#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define N_PIXELS 2296960
#define N_SECTORS 32

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__global__ void kernel(short *a, int offset, short *dark, int offsetDark, int *sectorSum)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  int iDark = offsetDark + threadIdx.x + blockIdx.x*blockDim.x;
  a[i] -= dark[iDark];

  // calculate sum per sector
  int iSector = ((offset / N_PIXELS) * N_SECTORS) + hfloor(iDark / N_SECTORS);
  sectorSum[iSector] = iSector;
  //atomicAdd(&sectorSum[mySector], mySector);
  //sectorSum[mySector] = mySector;
}

__global__ void common_mode(int *blockSum, int *sectorMean, int offsetSector)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  // calculate sector mean
  atomicAdd(&sectorMean[offsetSector], blockSum[i]);
}

__global__ void common_mode_apply(short *a, int offset, int *sectorMean, int offsetSector, int sectorSize)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  a[i] = a[i] - (sectorMean[offsetSector]/sectorSize);
}

/* ---------------------- host code -----------------------------*/
void fill( short *p, int n, int val ) {
  for(int i = 0; i < n; i++){
    p[i] = val;
  }
}

float maxError(short *a, int n)
{
  float maxE = 0;
  for (int i = 0; i < n; i++) {
    float error = fabs(a[i]-1.0f);
    if (error > maxE) maxE = error;
  }
  return maxE;
}

int main(int argc, char **argv)
{
  const int maxQuads = 4, maxSectors = 8;
  const int nColumns = 185, nRows = 388;
  const int nPixels = nColumns * nRows * maxSectors * maxQuads;
  const int nEvents = atoi(argv[1]);
  const int n = nPixels * nEvents;

  int nStreams = 16 * nEvents / 10;
  if (nStreams < 16) nStreams = 16;
  const int streamSize = n / nStreams;
  const int nSectors = maxQuads * maxSectors * nEvents;

  const int streamBytes = streamSize * sizeof(short);
  const int bytes = n * sizeof(short);
  const int darkBytes = nPixels * sizeof(short);
  const int sumSectorBytes = nSectors * sizeof(int);

  // a block has 1024 threads
  const int blockSize = 185;
  printf("Running with nStreams: %d streamSize: %d\n", nStreams, streamSize);
  int gridSize = streamSize / blockSize;
  printf("blockSize: %d gridSize: %d\n", blockSize, gridSize);

  int devId = 0;
  if (argc > 2) devId = atoi(argv[2]);
  
  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda( hipSetDevice(devId) );

  // allocate pinned host memory and device memory
  short *a, *d_a; // data
  checkCuda( hipHostMalloc((void**)&a, bytes) ); // host pinned
  checkCuda( hipMalloc((void**)&d_a, bytes) ); // device  
  short *dark, *d_dark; // dark
  checkCuda( hipHostMalloc((void**)&dark, darkBytes) ); 
  checkCuda( hipMalloc((void**)&d_dark, darkBytes) ); 
  int *d_sectorSum, *sectorSum; // sum of each sector
  checkCuda( hipMalloc((void**)&d_sectorSum, sumSectorBytes) ); 
  hipMemset(d_sectorSum, 0, sumSectorBytes);
  sectorSum = (int *) malloc(sumSectorBytes);
  
  // prepare raw and dark data
  fill(a, n, 3);
  fill(dark, nPixels, 1);
  printf("Input values (Data): %d %d %d...%d %d %d\n", a[0], a[1], a[2], a[n-3], a[n-2], a[n-1]);
  printf("Input values (Dark): %d %d %d...%d %d %d\n", dark[0], dark[1], dark[2], dark[nPixels-3], dark[nPixels-2], dark[nPixels-1]);

  // serial copy for one dark 
  checkCuda( hipMemcpy(d_dark, dark, darkBytes, hipMemcpyHostToDevice) );

  float ms; // elapsed time in milliseconds

  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  checkCuda( hipEventCreate(&dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );

  // asynchronous version 1: loop over {copy, kernel, copy}
  checkCuda( hipEventRecord(startEvent, 0) );
  hipProfilerStart();
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    int offsetDark = offset % nPixels;
    printf("Stream :%d offset:%d offsetDark:%d\n", i, offset, offsetDark);
    checkCuda( hipMemcpyAsync(&d_a[offset], &a[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]) );
    kernel<<<gridSize, blockSize, 0, stream[i]>>>(d_a, offset, d_dark, offsetDark, d_sectorSum);
    checkCuda( hipMemcpyAsync(&a[offset], &d_a[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]) );
  }
  
  hipMemcpy(sectorSum, d_sectorSum, sumSectorBytes, hipMemcpyDeviceToHost);
  for (int i =0; i< nEvents * N_SECTORS; i++){
    printf("i: %d, sectorSum[i]: %d \n", i, sectorSum[i]);
  }
  //printf("Output values: %d %d %d...%d %d %d\n", a[0], a[1], a[2], a[143559], a[143560], a[143561]);
  hipProfilerStop();
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));
  
  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipEventDestroy(dummyEvent) );
  for (int i = 0; i < nStreams; ++i)
    checkCuda( hipStreamDestroy(stream[i]) );
  hipFree(d_a);
  hipHostFree(a);
  //hipFree(d_dark);
  //hipHostFree(dark);

  return 0;
}
